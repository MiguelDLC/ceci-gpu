#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main.h"
#include <math.h>
#include <chrono>

__global__ void dudt(
  Array3D<scalar,3> solution,
  Array3D<int,2> neighbours,
  Array3D<scalar,_NF> data,
  scalar g,
  scalar rho_0,
  Array3D<scalar,3> rhs,
  int n_triangles
){

  int ie = blockIdx.x*blockDim.x + threadIdx.x;
  // int ie_loc = threadIdx.x;

  if(ie < n_triangles){
    auto soli = load_elem(solution, ie);
    LocalArray3D<3> rhsi(0);

    // edge terms
    for (int side = 0; side <3; ++side) {
      int tri_l = ie;
      int closure_l = side;
      int tri_r     = neighbours(0, side, tri_l);
      int closure_r = neighbours(1, side, tri_l);

      scalar length,nx,ny;
      scalar x0 = data(_X, side,       tri_l);
      scalar x1 = data(_X, (side+1)%3, tri_l);
      scalar y0 = data(_Y, side,       tri_l);
      scalar y1 = data(_Y, (side+1)%3, tri_l);
      length = hypot(x0-x1,y0-y1);
      nx = -(y0-y1)/length;
      ny = (x0-x1)/length;

      for (int iq = 0; iq < gauss_edge_n; ++iq) {
        scalar phil[3];
        scalar xitl[2];
        fe_2d_closure_xi(gauss_edge_xi[iq],closure_l,xitl);
        tri_phi_e(xitl,phil);

        scalar soll[3];
        soll[0] = fe_2d_interp_field(soli, 0, 0, phil);
        soll[1] = fe_2d_interp_field(soli, 1, 0, phil);
        soll[2] = fe_2d_interp_field(soli, 2, 0, phil);

        scalar bathl = fe_2d_interp_field(data, _BATH, tri_l, phil);
        scalar c_l   = fe_2d_interp_field(data, _C   , tri_l, phil);

        scalar flux[3] = {0};
        if (tri_r >= 0) {
          scalar phir[3];
          scalar xitr[2];
          fe_2d_closure_xi(gauss_edge_xi[iq],closure_r,xitr);
          tri_phi_e(xitr,phir);

          scalar solr[3];
          solr[0] = fe_2d_interp_field(solution, 0, tri_r, phir);
          solr[1] = fe_2d_interp_field(solution, 1, tri_r, phir);
          solr[2] = fe_2d_interp_field(solution, 2, tri_r, phir);

          scalar bathr = fe_2d_interp_field(data, _BATH, tri_r, phir);
          scalar c_r   = fe_2d_interp_field(data, _C   , tri_r, phir);

          iflux(c_l, c_r, soll, solr, bathl, bathr, nx, ny, g, flux);
        }

        scalar w = gauss_edge_w[iq]*(length/2);
        fe_2d_assemble_term(rhsi,3,3,phil,NULL,-w,flux);
      }
    }


    // volume term
    scalar dphi[3][2], jac;
    fe_2d_triangle(data,ie,dphi,&jac);

    for (int iq=0; iq < gauss_tri_n; ++iq) {
      scalar phi[3];
      tri_phi(gauss_tri_xi[iq],phi);
      
      scalar sol[3];
      sol[0] = fe_2d_interp_field(soli, 0, 0, phi);
      sol[1] = fe_2d_interp_field(soli, 1, 0, phi);
      sol[2] = fe_2d_interp_field(soli, 2, 0, phi);

      scalar dsol[3][2];
      fe_2d_grad_field(soli, 0, 0, dphi, dsol[0]);
      fe_2d_grad_field(soli, 1, 0, dphi, dsol[1]);
      fe_2d_grad_field(soli, 2, 0, dphi, dsol[2]);

      scalar bath, cor, tau[2], gamma;
      bath   = fe_2d_interp_field(data, _BATH , ie, phi);
      cor    = fe_2d_interp_field(data, _COR  , ie, phi);
      tau[0] = fe_2d_interp_field(data, _TAUX , ie, phi);
      tau[1] = fe_2d_interp_field(data, _TAUY , ie, phi);
      gamma  = fe_2d_interp_field(data, _GAMMA, ie, phi);
      
      scalar dbath[2];
      fe_2d_grad_field(data, _BATH, ie, dphi, dbath);

      scalar s[9] = {0};
      fvolume(sol, dsol, bath, dbath, cor, tau, s, g, rho_0, gamma);

      scalar w = gauss_tri_w[iq]*jac;
      fe_2d_assemble_term(rhsi,3,3,phi,dphi,w,s);
    }

    // inverse mass matrix
    fe_2d_multiply_inv_mass_matrix(jac,rhsi);

    // write back
    for (int field = 0; field <3; ++field) {
      for (int node = 0; node <3; ++node) {
        rhs(field, node, ie) = rhsi(field, node);
      }
    }
  }
}

__global__ void axpy(scalar* x, scalar* y, scalar* z, scalar a, scalar b, int n){
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < n){
    z[i] = a*x[i] + b*y[i];
  }
}

int main(){
  Array3D<scalar,_NF> data;
  Array3D<int,2> neighbours;
  int n_elem = readmesh("square.txt", data, neighbours);

  Array3D<scalar,3> solution;
  solution.data = malloc_manged_flags<scalar>(3*3*n_elem);
  solution.n_elem = n_elem;

  Array3D<scalar,3> solution_mid;
  solution_mid.data = malloc_manged_flags<scalar>(3*3*n_elem);
  solution_mid.n_elem = n_elem;

  Array3D<scalar,3> rhs;
  rhs.data = malloc_manged_flags<scalar>(3*3*n_elem);
  rhs.n_elem = n_elem;  

  scalar dt = 2.0;
  int ngrid_dudt = (n_elem+BLOCK_SIZE-1)/BLOCK_SIZE;
  int ngrid_axpy = (n_elem*9+BLOCK_SIZE-1)/BLOCK_SIZE;
  ERRCHK(hipDeviceSynchronize());

  auto tstart = std::chrono::high_resolution_clock::now();
  const int niter = 10;
  double ts[niter+1] = {0};
  for(int iter = 0; iter < niter; iter++){
    int nsub = 1000;
    for(int i = 0; i < nsub; i++){
      // classical RK2
      dudt<<<ngrid_dudt,BLOCK_SIZE>>>(solution, neighbours, data, 9.81, 1000, rhs, n_elem);
      axpy<<<ngrid_axpy,BLOCK_SIZE>>>(solution.data, rhs.data, solution_mid.data, 1, dt/2, n_elem*9);
      dudt<<<ngrid_dudt,BLOCK_SIZE>>>(solution_mid, neighbours, data, 9.81, 1000, rhs, n_elem);
      axpy<<<ngrid_axpy,BLOCK_SIZE>>>(solution.data, rhs.data, solution.data, 1, dt, n_elem*9);
    }
    ERRCHK(hipDeviceSynchronize());
    auto tstop = std::chrono::high_resolution_clock::now();
    ts[iter+1] = std::chrono::duration<double>(tstop-tstart).count()/nsub;
  }
  write_result("solution.txt", solution);
  double dt_min = ts[1];
  double dt_sum1 = 0;
  double dt_sum2 = 0;
  for(int i = 0; i < niter; i++){
    double dmt = ts[i+1]-ts[i];
    dt_sum1 += dmt;
    dt_sum2 += dmt*dmt;
    dt_min = fmin(dt_min, dmt);
  }
  double dt_mean = dt_sum1/niter;
  double dt_std = sqrt(dt_sum2/niter - dt_mean*dt_mean);
  printf("Min  time per iter: %.1f us\n", dt_min*1e6);
  printf("Mean time per iter: %.1f us\n", dt_mean*1e6);
  printf("Std : %.3f us\n", dt_std*1e6);
  return 0;
}

